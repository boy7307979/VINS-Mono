#include "hip/hip_runtime.h"
#include "sgm_impl.h"
#include <cassert>

static texture<float, hipTextureType2D, hipReadModeElementType> tex2d_ref;
static texture<float, hipTextureType2D, hipReadModeElementType> tex2d_cur;

#define INDEX(v, u, d, width, nd) ((v*width+u)*nd + d)

__global__ void SADCalcCostKernel(int meas_cnt, int num_disparity, float bf,
                                  float* H, float* h, size_t height, size_t width,
                                  float* cost) {
    // blockDim.x = num_disparity
    // gridDim.x = width
    // gridDim.y = height
    // blockIdx.x = [0, width - 1]
    // blockIdx.y = [0, height - 1]
    // threadIdx.x = [0, num_disparity - 1]
    const int p_u = blockIdx.x;
    const int p_v = blockIdx.y;
    const int disparity = threadIdx.x;

    // H = [H[0] H[1] H[2],
    //      H[3] H[4] H[5],
    //      H[6] H[7] H[8]]

    // h = [h[0] h[1] h[2]]'
    float Hp[3];
    Hp[0] = H[0] * p_u + H[1] * p_v + H[2];
    Hp[1] = H[3] * p_u + H[4] * p_v + H[5];
    Hp[2] = H[6] * p_u + H[7] * p_v + H[8];

    float Hpu[3]; // (0, -1)
    Hpu[0] = Hp[0] - H[1];
    Hpu[1] = Hp[1] - H[4];
    Hpu[2] = Hp[2] - H[7];

    float Hpd[3]; // (0, 1)
    Hpd[0] = Hp[0] + H[1];
    Hpd[1] = Hp[1] + H[4];
    Hpd[2] = Hp[2] + H[7];

    float Hpl[3]; // (-1, 0)
    Hpl[0] = Hp[0] - H[0];
    Hpl[1] = Hp[1] - H[3];
    Hpl[2] = Hp[2] - H[6];

    float Hpr[3]; // (1, 0)
    Hpr[0] = Hp[0] + H[0];
    Hpr[1] = Hp[1] + H[3];
    Hpr[2] = Hp[2] + H[6];

    float Hpul[3]; // (0, -1) + (-1, 0)
    Hpul[0] = Hpu[0] - H[0];
    Hpul[1] = Hpu[1] - H[3];
    Hpul[2] = Hpu[2] - H[6];

    float Hpur[3];
    Hpur[0] = Hpu[0] + H[0];
    Hpur[1] = Hpu[1] + H[3];
    Hpur[2] = Hpu[2] + H[6];

    float Hpdl[3];
    Hpdl[0] = Hpd[0] - H[0];
    Hpdl[1] = Hpd[1] - H[3];
    Hpdl[2] = Hpd[2] - H[6];

    float Hpdr[3];
    Hpdr[0] = Hpd[0] + H[0];
    Hpdr[1] = Hpd[1] + H[3];
    Hpdr[2] = Hpd[2] + H[6];

    float *cost_ptr = cost + INDEX(p_u, p_v, disparity, width, num_disparity);
    float last_cost = (meas_cnt == 1)? 0 : *cost_ptr;
    if(meas_cnt != 1 && last_cost < 0)
        return;

    float cost_value = 0.0f;
    float inv_depth = disparity / bf;
#define PROJECT(H, uu, vv) {\
            float z = H[2] + h[2] * inv_depth; \
            float ppu = (H[0] + h[0] * inv_depth) / z; \
            float ppv = (H[1] + h[1] * inv_depth) / z; \
            if(ppu < 0 || ppv < 0 || ppu >= width || ppv >= height) { \
                *cost_ptr = -1.0f; \
                return; \
            } \
            cost_value += fabs(tex2D(tex2d_ref, p_u + 0.5 + uu, p_v + 0.5 + vv) - \
                               tex2D(tex2d_cur, ppu + 0.5, ppv + 0.5)); \
        }
    PROJECT(Hp, 0, 0);
    PROJECT(Hpu, 0, -1);
    PROJECT(Hpd, 0, 1);
    PROJECT(Hpl, -1, 0);
    PROJECT(Hpr, 1, 0);
    PROJECT(Hpul, -1, -1);
    PROJECT(Hpur, 1, -1);
    PROJECT(Hpdl, -1, 1);
    PROJECT(Hpdr, 1, 1);
#undef PROJECT

    if(meas_cnt == 1)
        *cost_ptr = cost_value / 9.0f;
    else
        *cost_ptr = (last_cost * (meas_cnt - 1) + cost_value / 9.0f) / meas_cnt;
}

void SADCalcCost(int meas_cnt, int num_disparity, float bf,
                 float* h_H, float* h_h, float* img_ref, float* img_cur,
                 size_t height, size_t width, size_t step, float* cost) {
    hipUnbindTexture(tex2d_cur);
    hipUnbindTexture(tex2d_ref);

    dim3 grid = dim3(width, height);
    dim3 block = dim3(num_disparity);

    hipChannelFormatDesc ca_desc0 = hipCreateChannelDesc<float>();
    hipChannelFormatDesc ca_desc1 = hipCreateChannelDesc<float>();
    tex2d_ref.addressMode[0] = hipAddressModeBorder;
    tex2d_ref.addressMode[1] = hipAddressModeBorder;
    tex2d_ref.filterMode = hipFilterModeLinear;
    tex2d_ref.normalized = false;
    tex2d_cur.addressMode[0] = hipAddressModeBorder;
    tex2d_cur.addressMode[1] = hipAddressModeBorder;
    tex2d_cur.filterMode = hipFilterModeLinear;
    tex2d_cur.normalized = false;

    size_t offset = 0;
    hipBindTexture2D(&offset, tex2d_ref, img_ref, ca_desc0, width, height, step);
    assert(offset == 0);
    hipBindTexture2D(&offset, tex2d_cur, img_cur, ca_desc1, width, height, step);
    assert(offset == 0);

    float *d_H, *d_h;
    hipMalloc(&d_H, 9 * sizeof(float));
    hipMalloc(&d_h, 3 * sizeof(float));

    hipMemcpy(d_H, h_H, 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_h, h_h, 3 * sizeof(float), hipMemcpyHostToDevice);

    SADCalcCostKernel<<<grid, block>>>(meas_cnt, num_disparity, bf,
                                       d_H, d_h, height, width, cost);

    hipDeviceSynchronize();

    hipFree(d_H);
    hipFree(d_h);
}

__global__ void SGMCalcCostKernel(int num_disparity, size_t height, size_t width, int idx,
                                  int start, int dx, int dy, int end, float p1, float p2, float tau_so,
                                  float sgm_q1, float sgm_q2, float* sad_cost, float* sgm_cost) {
    int xy[2] = {(int)blockIdx.x, (int)blockIdx.x};
    xy[idx] = start;
    int p_u = xy[0], p_v = xy[1];
    int d = threadIdx.x;

    __shared__ float *block_shared_array;
    float* output_s = block_shared_array;
    float* output_min = output_s + num_disparity;
    float* input_s =  output_min + num_disparity;
    float* input_min = input_s + num_disparity;

    input_s[d] = input_min[d] = sad_cost[INDEX(p_v, p_u, d, width, num_disparity)];
    __syncthreads();
    // find input_s min
    for(int i = (num_disparity >> 1); i > 0; i = (i >> 1)) {
        if(d < i && d + i < num_disparity && input_min[d + i] < input_min[d])
            input_min[d] = input_min[d + 1];
        __syncthreads();
    }

    if(input_min[0] < 0.0f) {
        input_s[d] = 0.0f;
        sgm_cost[INDEX(p_v, p_u, d, width, num_disparity)] = input_s[d];
        output_s[d] = output_min[d] = input_s[d];
    }
    else {
        sgm_cost[INDEX(p_v, p_u, d, width, num_disparity)] += input_s[d];
        output_s[d] = output_min[d] = input_s[d];
    }
    xy[0] += dx;
    xy[1] += dy;

    for(int k = 1; k < end; ++k, xy[0] += dx, xy[1] += dy) {
        p_u = xy[0];
        p_v = xy[1];

        input_s[d] = input_min[d] = sad_cost[INDEX(p_v, p_u, d, width, num_disparity)];
        __syncthreads();

        for(int i = (num_disparity >> 1); i > 0; i = (i >> 1)) {
            if(d < i && d + i < num_disparity) {
                if(output_min[d + i] < output_min[d])
                    output_min[d] = output_min[d + i];
                if(input_min[d + i] < input_min[d])
                    input_min[d] = input_min[d + i];
                __syncthreads();
            }
        }

        if(input_min[0] < 0.0f) {
            input_s[d] = 0.0f;
            __syncthreads();
        }

        float G = fabs(tex2D(tex2d_ref, p_u + 0.5, p_v + 0.5) -
                       tex2D(tex2d_ref, p_u - dx + 0.5, p_v -dy + 0.5));
        float P1 = p1, P2 = p2;
        if(G <= tau_so) {
            P1 *= sgm_q1;
            P2 *= sgm_q2;
        }

        float cost = min(output_s[d], output_min[0] + P2);
        if(d - 1 >=0)
            cost = min(cost, output_s[d - 1]);
        if(d + 1 < num_disparity)
            cost = min(cost, output_s[d + 1]);

        float val = input_s[d] + cost - output_min[0];
        if(input_min[0] < 0.0f)
            sgm_cost[INDEX(p_v, p_u, d, width, num_disparity)] = 0.0;
        else
            sgm_cost[INDEX(p_v, p_u, d, width, num_disparity)] += val;

        output_min[d] = output_s[d] = val;
        __syncthreads();
    }
}

void SGM4PathCalcCost(float p1, float p2, float tau_so, float sgm_q1,
                      float sgm_q2, int num_disparity, size_t height,
                      size_t width, float* sad_cost, float* sgm_cost) {
    SGMCalcCostKernel<<<height, num_disparity,
            4 * num_disparity * sizeof (float)>>>(num_disparity, height, width,
                                                  0, 0, 1, 0, width, p1, p2,
                                                 tau_so, sgm_q1, sgm_q2, sad_cost,
                                                 sgm_cost);
    SGMCalcCostKernel<<<height, num_disparity,
            4 * num_disparity * sizeof (float)>>>(num_disparity, height, width, 0,
                                                  width - 1, -1, 0, width, p1, p2,
                                                 tau_so, sgm_q1, sgm_q2, sad_cost,
                                                 sgm_cost);
    SGMCalcCostKernel<<<width, num_disparity,
            4 * num_disparity * sizeof (float)>>>(num_disparity, height, width,
                                                  1, 0, 0, 1, height, p1, p2,
                                                 tau_so, sgm_q1, sgm_q2, sad_cost,
                                                 sgm_cost);
    SGMCalcCostKernel<<<width, num_disparity,
            4 * num_disparity * sizeof (float)>>>(num_disparity, height, width, 1,
                                                  height - 1, 0, -1, height, p1, p2,
                                                 tau_so, sgm_q1, sgm_q2, sad_cost,
                                                 sgm_cost);
    hipDeviceSynchronize();
}

__global__ void PostprocessingKernel(float* cost, size_t height, size_t width, int num_disparity,
                                     float* dis_mat, size_t dis_step) {
    const int p_u = blockIdx.x;
    const int p_v = blockIdx.y;
    const int d = threadIdx.x;
    __shared__ float* shared_memory;

    float* cost_ptr = cost + p_v * dis_step + p_u;

    float* c = shared_memory;
    float* c_min = shared_memory + num_disparity;
    float* c_idx = c_min + num_disparity;
    c[d] = c_min[d] = cost[INDEX(p_v, p_u, d, width, num_disparity)];
    c_idx[d] = d;
    __syncthreads();
    for(int i = (num_disparity >> 1); i > 0; i = (i >> 1)) {
        if(d < i && d + i < num_disparity && c_min[d + i] < c_min[d]) {
            c_min[d] = c_min[d + i];
            c_idx[d] = c_idx[d + i];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        float min_cost = c_min[0];
        int min_idx = c_idx[0];

        if(min_cost == 0 || min_idx == 0 || min_idx == num_disparity - 1
                || c[min_idx - 1] + c[min_idx + 1] < 2 * min_cost) {
            *cost_ptr = 0;
        }
        else {
            float cost_pre = c[min_idx - 1];
            float cost_post = c[min_idx + 1];
            float a = cost_pre + cost_post - 2.0f * min_cost;
            float b = cost_post - cost_pre;
            float subpixel_idx = min_idx - b/(2*a);
            *cost_ptr = subpixel_idx;
        }
    }
}

void Postprocessing(float* cost, size_t height, size_t width, int num_disparity,
                    float* dis_mat, size_t dis_step) {
    dim3 block(num_disparity);
    dim3 grid(width, height);

    PostprocessingKernel
    <<<grid, block, num_disparity * 3 * sizeof(float)>>>
    (cost, height, width, num_disparity, dis_mat, dis_step);

    hipDeviceSynchronize();
}
#undef INDEX
